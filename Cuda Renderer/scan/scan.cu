#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2. 
 */
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

void printArray(std::string s, int *device_data, int length) {
    printf("array[%s] (length=%d)\n", s.c_str(), length);
    int *host_data = (int *) malloc(length * sizeof(int));
    hipMemcpy(host_data, device_data, length * sizeof(int),
               hipMemcpyDeviceToHost);
    for (size_t i = 0; i < length; ++i) {
        printf("%d ", host_data[i]);
    }
    printf("\n");
}

__global__ void
scan_upsweep(int N, int *data, int twod1, int twod) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
        data[index * twod1 + twod1 - 1] += data[index * twod1 + twod - 1];
}

__global__ void
scan_downsweep(int N, int *data, int twod1, int twod) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        int t = data[index * twod1 + twod - 1];
        data[index * twod1 + twod - 1] = data[index * twod1 + twod1 - 1];
        data[index * twod1 + twod1 - 1] += t;
    }
}

__global__ void upsweep_ending_kernel(int *data, int length) {
    data[length - 1] = 0;
}


void exclusive_scan(int *device_data, int length) {
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA 
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */

    int calnum;
    length = nextPow2(length);
    //printArray("original data", device_data, length);
    for (int twod = 1; twod < length; twod *= 2) {
        int twod1 = twod * 2;
        calnum = length / twod1;
        // compute number of blocks and threads per block
        const int threadsPerBlock = 1024;
        const int blocks = (calnum + threadsPerBlock - 1) / threadsPerBlock;
        scan_upsweep<<<blocks, threadsPerBlock>>>(calnum, device_data, twod1, twod);
        hipDeviceSynchronize();
        //printArray("****round", device_data, length);
    }

    //printArray("after upsweep", device_data, length);
    upsweep_ending_kernel<<<1, 1>>>(device_data, length);

    for (int twod = length / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        calnum = length / twod1;
        const int threadsPerBlock = 1024;
        const int blocks = (calnum + threadsPerBlock - 1) / threadsPerBlock;
        scan_downsweep<<<blocks, threadsPerBlock>>>(calnum, device_data, twod1, twod);
        hipDeviceSynchronize();
//        printArray("down round***", device_data, length);
    }

    //printArray("after downsweep", device_data, length);
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int *inarray, int *end, int *resultarray) {
    int *device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness. 
    // You may have an easier time in your implementation if you assume the 
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    printf("*******initail lenght=%ld, rounded_length=%d\n", end - inarray, rounded_length);

    hipMalloc((void **) &device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int *inarray, int *end, int *resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_peaks_kernel(int *input, int length, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index == 0 || index == length - 1) {
        output[index] = 0;
        return;
    }

    if (input[index - 1] < input[index] && input[index + 1] < input[index])
        output[index] = 1;
    else
        output[index] = 0;
}

__global__ void update_output(int *input, int length, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0 || index == length - 1) {
        return;
    }
    if (input[index] != input[index + 1]) {
        output[input[index]] = index;
    }
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if 
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    int *isPeaks;
    hipMalloc(&isPeaks, sizeof(int) * length);

    const int threadsPerBlock = 1024;
    const int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    //printArray("This is device_input: ", device_input, length);

    find_peaks_kernel<<<blocks, threadsPerBlock>>>(device_input, length, isPeaks);
    hipDeviceSynchronize();
    //printArray("This is isPeaks: ", isPeaks, length);

    exclusive_scan(isPeaks, length);
    //printArray("This is peaks_Pre_sum: ", isPeaks, length);

    update_output<<<blocks, threadsPerBlock>>>(isPeaks, length, device_output);
    hipDeviceSynchronize();
    //printArray("This is output: ", device_output, length);


    int peakCnt;
    hipMemcpy(&peakCnt, &isPeaks[length-1], sizeof(int), hipMemcpyDeviceToHost);
    //printf("peakCnt is :%d!!!!!!!!!\n", peakCnt);
    hipFree(isPeaks);
    return peakCnt;
}


/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **) &device_input, rounded_length * sizeof(int));
    hipMalloc((void **) &device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo() {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
